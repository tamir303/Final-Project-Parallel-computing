#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"
#include "general.h"

__device__ double calcX(double x1, double x2, double t) {
    return ((x2 - x1) / 2) * sin(t * PI) + ((x2 + x1) / 2);
}

__device__ double calcY(double x, double a, double b) {
    return a * x + b;
}

/**
 * Calculates the coordinates of points based on input data and user-defined functions.
 *
 * @param cords         Pointer to an array of Cord objects representing the coordinates (x and y) of points at different times.
 */
__global__ void calcCords(Cord* cords, int pSize) {
    // Get the indices of the current thread and the total number of threads
    int tCount = blockIdx.x;
    int offset = pSize * tCount;
    int start = threadIdx.x * (pSize / blockDim.x);
    int end = blockDim.x - 1 == threadIdx.x ? pSize : start + pSize / blockDim.x;

    Point p;
    double t, xCord;
    for (int i = start; i < end; i++) {
        p = cords[i + offset].point;
        t = cords[i + offset].t;

        // Calculate the x coordinate using the user-defined function
        xCord = cords[i + offset].x = calcX(p.x1, p.x2, t);

        // Calculate the y coordinate using the user-defined function
        cords[i + offset].y = calcY(xCord, p.a, p.b);
    }
}

int calcCoordinates(Cord* cords, int pSize, int cSize) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t c_tSize = cSize * pSize * sizeof(Cord);
    int thread_num = 500;
    int block_num = cSize;

    // Allocate memory on GPU to copy the data from the host
    Cord *c_A;
    err = hipMalloc((void **)&c_A, c_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(c_A, cords, c_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    calcCords<<<block_num, thread_num>>>(c_A, pSize);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch calcCords kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cords, c_A, c_tSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from c_A to cords -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(c_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

// ################################################################################### //


__device__ int arePointsInDistance(double x1, double y1, double x2, double y2, double d) {
    double dx = x2 - x1;
    double dy = y2 - y1;
    return sqrt(dx * dx + dy * dy) < d;
}

/**
 * Calculates the coordinates of points based on input data and user-defined functions.
 *
 * @param cords         Pointer to an array of Cord objects representing the coordinates (x and y) of points at different times.
 */
__global__ void countPointsInDistance(Cord* cords, int* satisfiers, int pSize, double distance, int k) {
    // Get the indices of the current thread and the total number of threads
    int Pi, Pj, count;
    int tOffset = pSize * blockIdx.x;
    int start = tOffset + threadIdx.x * (pSize / blockDim.x);
    int end = blockDim.x - 1 == threadIdx.x ? tOffset + pSize : start + pSize / blockDim.x;

    for (Pi = start; Pi < end; Pi++) {
        count = 0;
        for (Pj = 0; Pj < pSize && count < k ; Pj++) {
            if (Pi != Pj) {
                Cord PiCords = cords[Pi];
                Cord PjCords = cords[Pj];
                count += arePointsInDistance(PiCords.x, PiCords.y, PjCords.x, PjCords.y, distance);
            }
        }

        satisfiers[Pi] = count >= k;
    }
}

__global__ void findFirstThreeOnes(const int* satisfiers, int* results, int* output, int pSize) {

    __shared__ int counter;
    if (threadIdx.x == 0)
        counter = -1;

    __syncthreads();

    int tOffset = pSize * blockIdx.x;
    int start = tOffset + threadIdx.x * (pSize / blockDim.x);
    int end = blockDim.x - 1 == threadIdx.x ? tOffset + pSize : start + pSize / blockDim.x;

    for (int i = start; i < end; i++) {
        if (satisfiers[i]) {
            int currIndex = atomicAdd(&counter, 1);
            if (currIndex < 3) {
                atomicExch(&output[blockIdx.x * 3 + currIndex], i % pSize);
            }
            else if (currIndex + 1 >= 3) {
                atomicExch(&results[blockIdx.x], 1);
                break;
            }
        }
    }

    __syncthreads();
}

int* calcProximityCriteria(Cord* cords, int tCount, double distance, int pSize, int k, int* output) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t c_tSize =  pSize * tCount * sizeof(Cord);
    size_t s_tSize = pSize * tCount * sizeof(int);
    int thread_num = 100;
    int block_num = tCount;

    // Allocate memory on GPU to copy the data from the host
    int *s_A;
    err = hipMalloc((void **)&s_A, s_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on s_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on GPU to copy the data from the host
    Cord *c_A;
    err = hipMalloc((void **)&c_A, c_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(c_A, cords, c_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    countPointsInDistance<<<block_num, thread_num>>>(c_A, s_A, pSize, distance, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch countPointsInDistance kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    int* d_output;
    int* d_results, *results = NULL;

    err = hipMalloc((void**)&d_output, 3 * tCount * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate d_output - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&d_results, tCount * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate d_output - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    findFirstThreeOnes<<<block_num, thread_num>>>(s_A, d_results, d_output, pSize);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch findFirstThreeOnes kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Copy data from host to the GPU memory
    results = (int*) allocateArray(tCount, sizeof(int));
    err = hipMemcpy(results, d_results, tCount * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(output, d_output, 3 * tCount * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

        // for (int i = 0; i < tCount; i++)
        // printf("%d %d %d res: %d\n", output[i * 3], output[i * 3 + 1], output[i * 3 +2], results[i]);

    // Free allocated memory on GPU
    if (hipFree(c_A) != hipSuccess) {
        fprintf(stderr, "Failed to free c_A- %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(s_A) != hipSuccess) {
        fprintf(stderr, "Failed to free s_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_output) != hipSuccess) {
        fprintf(stderr, "Failed to free d_output - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_results) != hipSuccess) {
        fprintf(stderr, "Failed to free d_output - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return results;
}
