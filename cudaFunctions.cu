#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"
#include "mathCalc.h"

__global__  void calcCords(int *cords) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    cords[i] = 
}


int computeOnGPU(Point* points, Cord* cords, int pSize, int cSize) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t p_tSize = pSize * sizeof(Point);
    size_t c_tSize = cSize * sizeof(Cord);
    int thread_num = pSize;
    int block_num = cSize;

    // Allocate memory on GPU to copy the data from the host
    Point *p_A;
    err = hipMalloc((void **)&p_A, p_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on p_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on GPU to copy the data from the host
    Cord *c_A;
    err = hipMalloc((void **)&c_A, c_tSize * pSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(p_A, points, p_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device p_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(c_A, cords, c_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    calcCords<<<block_num, thread_num>>>(c_A, p_A);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch calcCords kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cords, c_A, c_tSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from c_A to cords -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(c_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(p_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

