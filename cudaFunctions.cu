#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcX(double x1, double x2, double t) {
    return ((x2 - x1) / 2) * sin(t * PI) + ((x2 + x1) / 2);
}

__device__ double calcY(double x, double a, double b) {
    return a * x + b;
}

/**
 * Calculates the coordinates of points based on input data and user-defined functions.
 *
 * @param cords         Pointer to an array of Cord objects representing the coordinates (x and y) of points at different times.
 * @param points        Pointer to an array of Point objects containing information about initial positions and coefficients.S
 */
__global__ void calcCords(Cord* cords) {
    // Get the indices of the current thread and the total number of threads
    int pIndex =  threadIdx.x;
    int tCount = blockIdx.x;
    int offset = blockDim.x;

    Point p = cords[pIndex + tCount* offset].point;
    double t = cords[pIndex + tCount* offset].t;

    // Calculate the x coordinate using the user-defined function
    double xCord = cords[pIndex + tCount * offset].x = calcX(p.x1, p.x2, t);
    
    // Calculate the y coordinate using the user-defined function
    cords[pIndex + tCount * offset].y = calcY(xCord, p.a, p.b);
}



int computeOnGPU(Cord* cords, int pSize, int cSize) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t c_tSize = cSize * pSize * sizeof(Cord);
    int thread_num = pSize;
    int block_num = cSize;

    // Allocate memory on GPU to copy the data from the host
    Cord *c_A;
    err = hipMalloc((void **)&c_A, c_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(c_A, cords, c_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    calcCords<<<block_num, thread_num>>>(c_A);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch calcCords kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cords, c_A, c_tSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from c_A to cords -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(c_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

