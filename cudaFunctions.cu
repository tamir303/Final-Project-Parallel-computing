#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcX(double x1, double x2, double t) {
    return ((x2 - x1) / 2) * sin(t * PI) + ((x2 + x1) / 2);
}

__device__ double calcY(double x, double a, double b) {
    return a * x + b;
}

/**
 * Calculates the coordinates of points based on input data and user-defined functions.
 *
 * @param cords         Pointer to an array of Cord objects representing the coordinates (x and y) of points at different times.
 */
__global__ void calcCords(Cord* cords, int pSize) {
    // Get the indices of the current thread and the total number of threads
    int tCount = blockIdx.x;
    int offset = pSize * tCount;
    int start = threadIdx.x * (pSize / blockDim.x);
    int end = blockDim.x - 1 == threadIdx.x ? pSize : start + pSize / blockDim.x;

    Point p;
    double t, xCord;
    for (int i = start; i < end; i++) {
        p = cords[i + offset].point;
        t = cords[i + offset].t;

        // Calculate the x coordinate using the user-defined function
        xCord = cords[i + offset].x = calcX(p.x1, p.x2, t);

        // Calculate the y coordinate using the user-defined function
        cords[i + offset].y = calcY(xCord, p.a, p.b);
    }
}

int calcCoordinates(Cord* cords, int pSize, int cSize) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t c_tSize = cSize * pSize * sizeof(Cord);
    int thread_num = 100;
    int block_num = cSize;

    // Allocate memory on GPU to copy the data from the host
    Cord *c_A;
    err = hipMalloc((void **)&c_A, c_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(c_A, cords, c_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    calcCords<<<block_num, thread_num>>>(c_A, pSize);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch calcCords kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cords, c_A, c_tSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from c_A to cords -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(c_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

// ################################################################################### //


__device__ int arePointsInDistance(double x1, double y1, double x2, double y2, double d) {
    double dx = x2 - x1;
    double dy = y2 - y1;
    return sqrt(dx * dx + dy * dy) < d;
}

/**
 * Calculates the coordinates of points based on input data and user-defined functions.
 *
 * @param cords         Pointer to an array of Cord objects representing the coordinates (x and y) of points at different times.
 */
__global__ void countPointsInDistance(Cord* cords, int* satisfiers, int pSize, double distance, int k) {
    // Get the indices of the current thread and the total number of threads
    int Pi, Pj, count;
    int start = threadIdx.x * (pSize / blockDim.x);
    int end = blockDim.x - 1 == threadIdx.x ? pSize : start + pSize / blockDim.x;

    for (Pi = start; Pi < end; Pi++) {
        count = 0;
        for (Pj = 0; Pj < pSize && count < k ; Pj++) {
            if (Pi != Pj) {
                Cord PiCords = cords[Pi];
                Cord PjCords = cords[Pj];
                count += arePointsInDistance(PiCords.x, PiCords.y, PjCords.x, PjCords.y, distance);
            }
        }

        satisfiers[Pi] = count >= k;
        if (satisfiers[Pi]) printf("Point: %d\n", Pi);
    }
}

int* calcProximityCriteria(Cord* cords, double distance, int pSize, int k) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t c_tSize =  pSize * sizeof(Cord);
    size_t s_tSize = pSize * sizeof(int);
    int* satisfiers;
    int thread_num = 100;

    satisfiers = (int*) malloc(s_tSize);

    // Allocate memory on GPU to copy the data from the host
    int *s_A;
    err = hipMalloc((void **)&s_A, s_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on s_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on GPU to copy the data from the host
    Cord *c_A;
    err = hipMalloc((void **)&c_A, c_tSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory on c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(c_A, cords, c_tSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device c_A - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    countPointsInDistance<<<1, thread_num>>>(c_A, s_A, pSize, distance, k);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch countPointsInDistance kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(satisfiers, s_A, s_tSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from s_A to satisfiers -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(c_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(s_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return satisfiers;
}
